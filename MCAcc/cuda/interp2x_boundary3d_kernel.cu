#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

namespace {

template <typename scalar_t>
__global__ void interp2x_boundary3d_cuda_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,5> input,
    torch::PackedTensorAccessor32<scalar_t,5> output,
    torch::PackedTensorAccessor32<bool,5> is_boundary,
    const float balance_value) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    const int bn = output.size(0);
    const int c = output.size(1);
    const int d = output.size(2);
    const int h = output.size(3);
    const int w = output.size(4);

    if (i >= bn * c * d * h * w) {
        return;
    }
    
    const int x = i % w;
    const int y = (i / w) % h;
    const int z = (i / (h * w)) % d;
    const int ci = (i / (d * h * w)) % c;
    const int bi = i / (c * d * h * w);

    const bool skip_x = x % 2 == 0;
    const bool skip_y = y % 2 == 0;
    const bool skip_z = z % 2 == 0; 

    if (skip_x && skip_y && skip_z){
        output[bi][ci][z][y][x] = input[bi][ci][z/2][y/2][x/2];
        is_boundary[bi][ci][z][y][x] = false;
        return;

    }else if (skip_x && skip_z){
        auto v1 = input[bi][ci][z/2][(y-1)/2][x/2];
        auto v2 = input[bi][ci][z/2][(y+1)/2][x/2];
        output[bi][ci][z][y][x] = (v1 + v2) / 2.;

        bool flag1 = v1 > balance_value;
        bool flag2 = v2 > balance_value;
        if (flag1 == flag2){is_boundary[bi][ci][z][y][x] = false;}
        else{is_boundary[bi][ci][z][y][x] = true;}
        return;

    }else if (skip_y && skip_z){
        auto v1 = input[bi][ci][z/2][y/2][(x-1)/2];
        auto v2 = input[bi][ci][z/2][y/2][(x+1)/2];
        output[bi][ci][z][y][x] = (v1 + v2) / 2.;

        bool flag1 = v1 > balance_value;
        bool flag2 = v2 > balance_value;
        if (flag1 == flag2){is_boundary[bi][ci][z][y][x] = false;}
        else{is_boundary[bi][ci][z][y][x] = true;}
        return;

    }else if (skip_x && skip_y){
        auto v1 = input[bi][ci][(z-1)/2][y/2][x/2];
        auto v2 = input[bi][ci][(z+1)/2][y/2][x/2];
        output[bi][ci][z][y][x] = (v1 + v2) / 2.;

        bool flag1 = v1 > balance_value;
        bool flag2 = v2 > balance_value;
        if (flag1 == flag2){is_boundary[bi][ci][z][y][x] = false;}
        else{is_boundary[bi][ci][z][y][x] = true;}
        return;

    }else if (skip_z){
        auto v1 = input[bi][ci][z/2][(y-1)/2][(x-1)/2];
        auto v2 = input[bi][ci][z/2][(y-1)/2][(x+1)/2]; 
        auto v3 = input[bi][ci][z/2][(y+1)/2][(x-1)/2]; 
        auto v4 = input[bi][ci][z/2][(y+1)/2][(x+1)/2];
        output[bi][ci][z][y][x] = (v1 + v2 + v3 + v4) / 4.0;

        bool flag1 = v1 > balance_value;
        bool flag2 = v2 > balance_value;
        bool flag3 = v3 > balance_value;
        bool flag4 = v4 > balance_value;
        if (flag1 == flag2 && flag2 == flag3 && flag3 == flag4){
            is_boundary[bi][ci][z][y][x] = false;
        }else{is_boundary[bi][ci][z][y][x] = true;}
        return;

    }else if (skip_x){
        auto v1 = input[bi][ci][(z-1)/2][(y-1)/2][x/2];
        auto v2 = input[bi][ci][(z+1)/2][(y-1)/2][x/2]; 
        auto v3 = input[bi][ci][(z-1)/2][(y+1)/2][x/2]; 
        auto v4 = input[bi][ci][(z+1)/2][(y+1)/2][x/2];
        output[bi][ci][z][y][x] = (v1 + v2 + v3 + v4) / 4.0;

        bool flag1 = v1 > balance_value;
        bool flag2 = v2 > balance_value;
        bool flag3 = v3 > balance_value;
        bool flag4 = v4 > balance_value;
        if (flag1 == flag2 && flag2 == flag3 && flag3 == flag4){
            is_boundary[bi][ci][z][y][x] = false;
        }else{is_boundary[bi][ci][z][y][x] = true;}
        return;

    }else if (skip_y){
        auto v1 = input[bi][ci][(z-1)/2][y/2][(x-1)/2];
        auto v2 = input[bi][ci][(z+1)/2][y/2][(x-1)/2]; 
        auto v3 = input[bi][ci][(z-1)/2][y/2][(x+1)/2]; 
        auto v4 = input[bi][ci][(z+1)/2][y/2][(x+1)/2];
        output[bi][ci][z][y][x] = (v1 + v2 + v3 + v4) / 4.0;

        bool flag1 = v1 > balance_value;
        bool flag2 = v2 > balance_value;
        bool flag3 = v3 > balance_value;
        bool flag4 = v4 > balance_value;
        if (flag1 == flag2 && flag2 == flag3 && flag3 == flag4){
            is_boundary[bi][ci][z][y][x] = false;
        }else{is_boundary[bi][ci][z][y][x] = true;}
        return;
    
    }else{
        auto v1 = input[bi][ci][(z-1)/2][(y-1)/2][(x-1)/2];
        auto v2 = input[bi][ci][(z-1)/2][(y-1)/2][(x+1)/2];
        auto v3 = input[bi][ci][(z-1)/2][(y+1)/2][(x-1)/2];
        auto v4 = input[bi][ci][(z-1)/2][(y+1)/2][(x+1)/2];
        auto v5 = input[bi][ci][(z+1)/2][(y-1)/2][(x-1)/2];
        auto v6 = input[bi][ci][(z+1)/2][(y-1)/2][(x+1)/2];
        auto v7 = input[bi][ci][(z+1)/2][(y+1)/2][(x-1)/2];
        auto v8 = input[bi][ci][(z+1)/2][(y+1)/2][(x+1)/2];
        output[bi][ci][z][y][x] = (v1 + v2 + v3 + v4 + v5 + v6 + v7 + v8) / 8.0;

        bool flag1 = v1 > balance_value;
        bool flag2 = v2 > balance_value;
        bool flag3 = v3 > balance_value;
        bool flag4 = v4 > balance_value;
        bool flag5 = v5 > balance_value;
        bool flag6 = v6 > balance_value;
        bool flag7 = v7 > balance_value;
        bool flag8 = v8 > balance_value;
        if (flag1 == flag2 && flag2 == flag3 && flag3 == flag4 && 
            flag4 == flag5 && flag5 == flag6 && flag6 == flag7 && 
            flag7 == flag8){
            is_boundary[bi][ci][z][y][x] = false;
        }else{is_boundary[bi][ci][z][y][x] = true;}
        return;

    }
}


template <typename scalar_t>
__global__ void interp2x_boundary3d_cuda_backward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,5> grad_output,
    torch::PackedTensorAccessor32<scalar_t,5> grad_input) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    const int bn = grad_input.size(0);
    const int c = grad_input.size(1);
    const int d = grad_input.size(2);
    const int h = grad_input.size(3);
    const int w = grad_input.size(4);

    if (i >= bn * c * d * h * w) {
        return;
    }
    
    const int x = i % w;
    const int y = (i / w) % h;
    const int z = (i / (h * w)) % d;
    const int ci = (i / (d * h * w)) % c;
    const int bi = i / (c * d * h * w);

    auto grad = grad_output[bi][ci][z*2][y*2][x*2];

    // 6 edges
    if (x > 0){
        grad += grad_output[bi][ci][z*2][y*2][x*2 - 1] / 2.0;}
    if (x < w - 1){
        grad += grad_output[bi][ci][z*2][y*2][x*2 + 1] / 2.0;}
    if (y > 0){
        grad += grad_output[bi][ci][z*2][y*2 - 1][x*2] / 2.0;}
    if (y < h - 1){
        grad += grad_output[bi][ci][z*2][y*2 + 1][x*2] / 2.0;}
    if (z > 0){
        grad += grad_output[bi][ci][z*2 - 1][y*2][x*2] / 2.0;}
    if (z < d - 1){
        grad += grad_output[bi][ci][z*2 + 1][y*2][x*2] / 2.0;}
    
    // 12 faces: xy
    if (x > 0 && y > 0){
        grad += grad_output[bi][ci][z*2][y*2 - 1][x*2 - 1] / 4.0;}
    if (x < w - 1 && y > 0){
        grad += grad_output[bi][ci][z*2][y*2 - 1][x*2 + 1] / 4.0;}
    if (x > 0 && y < h - 1){
        grad += grad_output[bi][ci][z*2][y*2 + 1][x*2 - 1] / 4.0;}
    if (x < w - 1 && y < h - 1){
        grad += grad_output[bi][ci][z*2][y*2 + 1][x*2 + 1] / 4.0;}
    // 12 faces: xz
    if (x > 0 && z > 0){
        grad += grad_output[bi][ci][z*2 - 1][y*2][x*2 - 1] / 4.0;}
    if (x < w - 1 && z > 0){
        grad += grad_output[bi][ci][z*2 - 1][y*2][x*2 + 1] / 4.0;}
    if (x > 0 && z < d - 1){
        grad += grad_output[bi][ci][z*2 + 1][y*2][x*2 - 1] / 4.0;}
    if (x < w - 1 && z < d - 1){
        grad += grad_output[bi][ci][z*2 + 1][y*2][x*2 + 1] / 4.0;}
    // 12 faces: yz
    if (y > 0 && z > 0){
        grad += grad_output[bi][ci][z*2 - 1][y*2 - 1][x*2] / 4.0;}
    if (y < h - 1 && z > 0){
        grad += grad_output[bi][ci][z*2 - 1][y*2 + 1][x*2] / 4.0;}
    if (y > 0 && z < d - 1){
        grad += grad_output[bi][ci][z*2 + 1][y*2 - 1][x*2] / 4.0;}
    if (y < h - 1 && z < d - 1){
        grad += grad_output[bi][ci][z*2 + 1][y*2 + 1][x*2] / 4.0;}

    // 8 corners
    if (x > 0 && y > 0 && z > 0){
        grad += grad_output[bi][ci][z*2 - 1][y*2 - 1][x*2 - 1] / 8.0;}
    if (x < w - 1 && y > 0 && z > 0){
        grad += grad_output[bi][ci][z*2 - 1][y*2 - 1][x*2 + 1] / 8.0;}
    if (x > 0 && y < h - 1 && z > 0){
        grad += grad_output[bi][ci][z*2 - 1][y*2 + 1][x*2 - 1] / 8.0;}
    if (x < w - 1 && y < h - 1 && z > 0){
        grad += grad_output[bi][ci][z*2 - 1][y*2 + 1][x*2 + 1] / 8.0;}
    if (x > 0 && y > 0 && z < d - 1){
        grad += grad_output[bi][ci][z*2 + 1][y*2 - 1][x*2 - 1] / 8.0;}
    if (x < w - 1 && y > 0 && z < d - 1){
        grad += grad_output[bi][ci][z*2 + 1][y*2 - 1][x*2 + 1] / 8.0;}
    if (x > 0 && y < h - 1 && z < d - 1){
        grad += grad_output[bi][ci][z*2 + 1][y*2 + 1][x*2 - 1] / 8.0;}
    if (x < w - 1 && y < h - 1 && z < d - 1){
        grad += grad_output[bi][ci][z*2 + 1][y*2 + 1][x*2 + 1] / 8.0;}
    
    grad_input[bi][ci][z][y][x] = grad;
    }
} // namespace


std::vector<torch::Tensor> interp2x_boundary3d_cuda_forward(
    const torch::Tensor& input, 
    const float balance_value) {
    
    torch::Device device = input.device();
    int bn = input.size(0);
    int c = input.size(1);
    int d = input.size(2) * 2 - 1;
    int h = input.size(3) * 2 - 1;
    int w = input.size(4) * 2 - 1;
    
    auto option1 = torch::TensorOptions().dtype(input.scalar_type()).device(device);
    auto output = torch::empty({bn, c, d, h, w}, option1);

    auto option2 = torch::TensorOptions().dtype(torch::ScalarType::Bool).device(device);
    auto is_boundary = torch::empty({bn, c, d, h, w}, option2);

    const int num_kernels = bn * c * d * h * w;
    const int num_threads = 1024;
    const dim3 blocks((num_kernels + num_threads - 1) / num_threads);

    AT_DISPATCH_FLOATING_TYPES(
        output.scalar_type(), "interp2x_boundary3d_cuda_forward", ([&] {
            interp2x_boundary3d_cuda_forward_kernel<scalar_t>
                <<<blocks, num_threads>>>(
                    input.packed_accessor32<scalar_t, 5>(), 
                    output.packed_accessor32<scalar_t, 5>(),
                    is_boundary.packed_accessor32<bool, 5>(),
                    balance_value);
    }));

    return {output, is_boundary};
}


torch::Tensor interp2x_boundary3d_cuda_backward(
    const torch::Tensor& grad_output) {
    
    torch::Device device = grad_output.device();
    int bn = grad_output.size(0);
    int c = grad_output.size(1);
    int d = (grad_output.size(2) + 1) / 2;
    int h = (grad_output.size(3) + 1) / 2;
    int w = (grad_output.size(4) + 1) / 2;

    auto option = torch::TensorOptions().dtype(grad_output.scalar_type()).device(device);
    auto grad_input = torch::empty({bn, c, d, h, w}, option);
    
    const int num_kernels = bn * c * d * h * w;
    const int num_threads = 1024;
    const dim3 blocks((num_kernels + num_threads - 1) / num_threads);

    AT_DISPATCH_FLOATING_TYPES(
        grad_output.scalar_type(), "interp2x_boundary3d_cuda_backward", ([&] {
            interp2x_boundary3d_cuda_backward_kernel<scalar_t>
                <<<blocks, num_threads>>>(
                    grad_output.packed_accessor32<scalar_t, 5>(), 
                    grad_input.packed_accessor32<scalar_t, 5>());
    }));

    return grad_input;
}

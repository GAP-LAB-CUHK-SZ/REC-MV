#include "hip/hip_runtime.h"
#include <memory>
#include "cuda_inc.h"
#include "CudaKernels.h"
static const float a2fVertexOffset[8][3] =
{
    {0.0, 0.0, 0.0},{1.0, 0.0, 0.0},{1.0, 1.0, 0.0},{0.0, 1.0, 0.0},
    {0.0, 0.0, 1.0},{1.0, 0.0, 1.0},{1.0, 1.0, 1.0},{0.0, 1.0, 1.0}
};
static const int a2iEdgeConnection[12][2] =
{
    {0,1}, {1,2}, {2,3}, {3,0},
    {4,5}, {5,6}, {6,7}, {7,4},
    {0,4}, {1,5}, {2,6}, {3,7}
};
static const float a2fEdgeDirection[12][3] =
{
    {1.0, 0.0, 0.0},{0.0, 1.0, 0.0},{-1.0, 0.0, 0.0},{0.0, -1.0, 0.0},
    {1.0, 0.0, 0.0},{0.0, 1.0, 0.0},{-1.0, 0.0, 0.0},{0.0, -1.0, 0.0},
    {0.0, 0.0, 1.0},{0.0, 0.0, 1.0},{ 0.0, 0.0, 1.0},{0.0,  0.0, 1.0}
};
static const int aiCubeEdgeFlags[256]=
{
    0x000, 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c, 0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
    0x190, 0x099, 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c, 0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
    0x230, 0x339, 0x033, 0x13a, 0x636, 0x73f, 0x435, 0x53c, 0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
    0x3a0, 0x2a9, 0x1a3, 0x0aa, 0x7a6, 0x6af, 0x5a5, 0x4ac, 0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
    0x460, 0x569, 0x663, 0x76a, 0x066, 0x16f, 0x265, 0x36c, 0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
    0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0x0ff, 0x3f5, 0x2fc, 0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
    0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x055, 0x15c, 0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
    0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0x0cc, 0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
    0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc, 0x0cc, 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
    0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c, 0x15c, 0x055, 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
    0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc, 0x2fc, 0x3f5, 0x0ff, 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
    0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c, 0x36c, 0x265, 0x16f, 0x066, 0x76a, 0x663, 0x569, 0x460,
    0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac, 0x4ac, 0x5a5, 0x6af, 0x7a6, 0x0aa, 0x1a3, 0x2a9, 0x3a0,
    0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c, 0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x033, 0x339, 0x230,
    0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c, 0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x099, 0x190,
    0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c, 0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x000
};
static const int a2iTriangleConnectionTable[256][16] =
{
    {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1},
    {3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1},
    {3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1},
    {3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1},
    {9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1},
    {9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
    {2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1},
    {8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1},
    {9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
    {4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1},
    {3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1},
    {1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1},
    {4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1},
    {4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1},
    {9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
    {5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1},
    {2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1},
    {9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
    {0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
    {2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1},
    {10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1},
    {4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1},
    {5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1},
    {5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1},
    {9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1},
    {0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1},
    {1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1},
    {10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1},
    {8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1},
    {2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1},
    {7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1},
    {9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1},
    {2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1},
    {11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1},
    {9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1},
    {5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1},
    {11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1},
    {11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
    {1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1},
    {9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1},
    {5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1},
    {2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
    {0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
    {5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1},
    {6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1},
    {3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1},
    {6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1},
    {5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1},
    {1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
    {10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1},
    {6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1},
    {8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1},
    {7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1},
    {3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
    {5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1},
    {0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1},
    {9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1},
    {8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1},
    {5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1},
    {0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1},
    {6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1},
    {10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1},
    {10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1},
    {8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1},
    {1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1},
    {3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1},
    {0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1},
    {10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1},
    {3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1},
    {6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1},
    {9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1},
    {8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1},
    {3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1},
    {6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1},
    {0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1},
    {10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1},
    {10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1},
    {2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1},
    {7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1},
    {7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1},
    {2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1},
    {1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1},
    {11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1},
    {8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1},
    {0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1},
    {7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
    {10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
    {2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
    {6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1},
    {7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1},
    {2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1},
    {1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1},
    {10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1},
    {10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1},
    {0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1},
    {7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1},
    {6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1},
    {8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1},
    {9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1},
    {6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1},
    {4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1},
    {10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1},
    {8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1},
    {0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1},
    {1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1},
    {8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1},
    {10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1},
    {4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1},
    {10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
    {5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
    {11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1},
    {9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
    {6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1},
    {7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1},
    {3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1},
    {7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1},
    {9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1},
    {3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1},
    {6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1},
    {9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1},
    {1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1},
    {4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1},
    {7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1},
    {6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1},
    {3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1},
    {0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1},
    {6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1},
    {0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1},
    {11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1},
    {6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1},
    {5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1},
    {9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1},
    {1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1},
    {1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1},
    {10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1},
    {0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1},
    {5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1},
    {10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1},
    {11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1},
    {9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1},
    {7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1},
    {2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1},
    {8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1},
    {9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1},
    {9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1},
    {1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1},
    {9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1},
    {9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1},
    {5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1},
    {0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1},
    {10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1},
    {2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1},
    {0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1},
    {0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1},
    {9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1},
    {5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1},
    {3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1},
    {5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1},
    {8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1},
    {0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1},
    {9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1},
    {1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1},
    {3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1},
    {4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1},
    {9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1},
    {11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1},
    {11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1},
    {2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1},
    {9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1},
    {3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1},
    {1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1},
    {4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1},
    {4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1},
    {0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1},
    {3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1},
    {3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1},
    {0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1},
    {9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1},
    {1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1}
};





inline __host__ __device__ float d_fGetOffset(float fValue1, float fValue2, float fValueDesired)
{
    double fDelta = fValue2 - fValue1;

    if(fDelta == 0.0)
    {
        return 0.5;
    }
    return (fValueDesired - fValue1)/fDelta;
}


__global__ void d_mc_get_mesh_on_gpu(int nthread, float* d_sdf, int *d_edge_point_state, int NX, int NY, int NZ, float fTargetValue, int *d_number_record, float* d_a2fVertexOffset, int *d_a2iEdgeConnection, float *d_a2fEdgeDirection, int* d_aiCubeEdgeFlags, int *d_a2iTriangleConnectionTable, float *d_points_coor, int *d_faces_ijkd, long int *d_faces_index)
{
    CUDA_KERNEL_LOOP(index, nthread)
    {
        int i,j,k;
        i=index/(NY*NZ);
        j=(index-i*NY*NZ)/NZ;
        k=index-i*NY*NZ-j*NZ;
        if( i<NX-1 && j<NY-1 && k<NZ-1)
//        if(true)
        {
            float fX, fY, fZ;
            fX = i;
            fY = j;
            fZ = k;
            
            int iCorner, iVertex, iVertexTest, iEdge, iTriangle, iFlagIndex, iEdgeFlags;
            float fOffset;
            float afCubeValue[8];
            float asEdgeVertex[12][3];
            
            for(iVertex = 0; iVertex < 8; iVertex++)
            {
                int index_to_use=(int)(fX + d_a2fVertexOffset[iVertex*3+0])*NY*NZ+(int)(fY + d_a2fVertexOffset[iVertex*3+1])*NZ+(int)(fZ + d_a2fVertexOffset[iVertex*3+2]);
                afCubeValue[iVertex] =d_sdf[index_to_use];
            }
            
            iFlagIndex = 0;
            for(iVertexTest = 0; iVertexTest < 8; iVertexTest++)
            {
                if(afCubeValue[iVertexTest] < fTargetValue)
                {
                    iFlagIndex |= 1<<iVertexTest;
                }
            }

            iEdgeFlags = d_aiCubeEdgeFlags[iFlagIndex];
            
            
            
            if(iEdgeFlags != 0)
            {
                
                for(iEdge = 0; iEdge < 12; iEdge++)
                {
                    if(iEdgeFlags & (1<<iEdge))
                    {
                        fOffset = d_fGetOffset(afCubeValue[ d_a2iEdgeConnection[iEdge*2+0] ], afCubeValue[ d_a2iEdgeConnection[iEdge*2+1] ], fTargetValue);
                        asEdgeVertex[iEdge][0] = fX + (d_a2fVertexOffset[d_a2iEdgeConnection[iEdge*2+0]*3+0] + fOffset * d_a2fEdgeDirection[iEdge*3+0]);
                        asEdgeVertex[iEdge][1] = fY + (d_a2fVertexOffset[d_a2iEdgeConnection[iEdge*2+0]*3+1] + fOffset * d_a2fEdgeDirection[iEdge*3+1]);
                        asEdgeVertex[iEdge][2] = fZ + (d_a2fVertexOffset[d_a2iEdgeConnection[iEdge*2+0]*3+2] + fOffset * d_a2fEdgeDirection[iEdge*3+2]);
                    }
                }
            
            
            
                bool is_ivertex_new[12];
                for (int iin =0;iin<12;iin++)
                {
                    is_ivertex_new[iin]=true;
                }
                for(iTriangle = 0; iTriangle < 5; iTriangle++)
                {
                    
                    if(d_a2iTriangleConnectionTable[iFlagIndex*16+3*iTriangle] < 0)
                    {
                        break;
                    }
                
                    int face_id = atomicAdd(&d_number_record[1],1);

                    for(iCorner = 0; iCorner < 3; iCorner++)
                    {
                        iVertex = d_a2iTriangleConnectionTable[iFlagIndex*16+3*iTriangle+iCorner];
                        int basex,basey,basez;
                        int direction;
                        if(iVertex==0)
                        {
                            basex=fX;
                            basey=fY;
                            basez=fZ;
                            direction=0;
                        }else if(iVertex==1)
                        {
                            basex=fX+1;
                            basey=fY;
                            basez=fZ;
                            direction=1;
                        }else if(iVertex==2)
                        {
                            basex=fX;
                            basey=fY+1;
                            basez=fZ;
                            direction=0;
                        }else if(iVertex==3)
                        {
                            basex=fX;
                            basey=fY;
                            basez=fZ;
                            direction=1;
                        }else if(iVertex==4)
                        {
                            basex=fX;
                            basey=fY;
                            basez=fZ+1;
                            direction=0;
                        }else if(iVertex==5)
                        {
                            basex=fX+1;
                            basey=fY;
                            basez=fZ+1;
                            direction=1;
                        }else if(iVertex==6)
                        {
                            basex=fX;
                            basey=fY+1;
                            basez=fZ+1;
                            direction=0;
                        }else if(iVertex==7)
                        {
                            basex=fX;
                            basey=fY;
                            basez=fZ+1;
                            direction=1;
                        }else if(iVertex==8)
                        {
                            basex=fX;
                            basey=fY;
                            basez=fZ;
                            direction=2;
                        }else if(iVertex==9)
                        {
                            basex=fX+1;
                            basey=fY;
                            basez=fZ;
                            direction=2;
                        }else if(iVertex==10)
                        {
                            basex=fX+1;
                            basey=fY+1;
                            basez=fZ;
                            direction=2;
                        }else if(iVertex==11)
                        {
                            basex=fX;
                            basey=fY+1;
                            basez=fZ;
                            direction=2;
                        }
                        
                        int vert_id=-1;
                        if (is_ivertex_new[iVertex]==false)
                        {
//                            vert_id = atomicMax(&d_edge_point_state[basex*NY*NZ*3+basey*NZ*3+basez*3+direction],-10);
                        }
                        else if (iVertex==0 || iVertex==3 || iVertex==8)
                        {
                            vert_id = atomicAdd(&d_number_record[0],1);
                            d_points_coor[vert_id*3+0]=asEdgeVertex[iVertex][0];
                            d_points_coor[vert_id*3+1]=asEdgeVertex[iVertex][1];
                            d_points_coor[vert_id*3+2]=asEdgeVertex[iVertex][2];
                            atomicExch(&d_edge_point_state[basex*NY*NZ*3+basey*NZ*3+basez*3+direction],vert_id);
                            is_ivertex_new[iVertex]=false;
                        }
                        d_faces_ijkd[face_id*3*4+iCorner*4+0]=basex;
                        d_faces_ijkd[face_id*3*4+iCorner*4+1]=basey;
                        d_faces_ijkd[face_id*3*4+iCorner*4+2]=basez;
                        d_faces_ijkd[face_id*3*4+iCorner*4+3]=direction;
                    }
                }
            }
        }
    }
}


__global__ void d_conver_ijkd_to_pindex(int nthread, int NX, int NY, int NZ, int *d_edge_point_state, int *d_faces_ijkd, long int *d_faces_index, int *d_number_record)
{
    CUDA_KERNEL_LOOP(index, nthread)
    {
        for (int pid = 0; pid<3; pid++)
        {
            int i = d_faces_ijkd[index*3*4+pid*4+0];
            int j = d_faces_ijkd[index*3*4+pid*4+1];
            int k = d_faces_ijkd[index*3*4+pid*4+2];
            int d = d_faces_ijkd[index*3*4+pid*4+3];
            d_faces_index[index*3+(2-pid)]=(long int)d_edge_point_state[i*NY*NZ*3+j*NZ*3+k*3+d];
        }
    }
}
__global__ void d_set_int(int nthread, int val, int* tar)
{
	CUDA_KERNEL_LOOP(index,nthread)
	{
		tar[index]=val;
	}
}
__global__ void d_scale_vertices(int nthread, float xstep,float ystep,float zstep,float xmin,float ymin,float zmin, float* vertices)
{
	CUDA_KERNEL_LOOP(index,nthread)
	{
		vertices[3*index]=vertices[3*index]*xstep+xmin;
		vertices[3*index+1]=vertices[3*index+1]*ystep+ymin;
		vertices[3*index+2]=vertices[3*index+2]*zstep+zmin;
	}
}

// support 8 gpus now
static std::vector<std::shared_ptr<MCGpu> > instances_=std::vector<std::shared_ptr<MCGpu> >(8,NULL);
MCGpu& MCGpu::Get(int device_id) {
  if (device_id<0||device_id>=8)
  	device_id=0;
  if (!instances_[device_id].get()) {
    instances_[device_id].reset(new MCGpu(device_id));
  }
  return *(instances_[device_id].get());
}

MCGpu::MCGpu(int device_id)
{
	device_id_=device_id;
	hipSetDevice(device_id_);
	hipMalloc((float**)&d_a2fVertexOffset_,sizeof(float)*8*3);
	hipMemcpy(d_a2fVertexOffset_,a2fVertexOffset,sizeof(float)*8*3,hipMemcpyHostToDevice);

    hipMalloc((int**)&d_a2iEdgeConnection_,sizeof(int)*12*2);
    hipMemcpy(d_a2iEdgeConnection_,a2iEdgeConnection,sizeof(int)*12*2,hipMemcpyHostToDevice);
    
    
    hipMalloc((float**)&d_a2fEdgeDirection_,sizeof(float)*12*3);
    hipMemcpy(d_a2fEdgeDirection_,a2fEdgeDirection,sizeof(float)*12*3,hipMemcpyHostToDevice);
    
    
    hipMalloc((int**)&d_aiCubeEdgeFlags_,sizeof(int)*256);
    hipMemcpy(d_aiCubeEdgeFlags_,aiCubeEdgeFlags,sizeof(int)*256,hipMemcpyHostToDevice);
    
    
    hipMalloc((int**)&d_a2iTriangleConnectionTable_,sizeof(int)*256*16);
    hipMemcpy(d_a2iTriangleConnectionTable_,a2iTriangleConnectionTable,sizeof(int)*256*16,hipMemcpyHostToDevice);

    hipMalloc((int**)&d_number_record_,sizeof(int)*2);
    hipMemset(d_number_record_,0,sizeof(int)*2);

    number_record_=std::vector<int>(2,0);

    NX=0;
    NY=0;
    NZ=0;
    mx=0;
    my=0;
    mz=0;
    d_edge_point_state_=NULL;
    d_points_coor_=NULL;
    d_faces_index_=NULL;
    d_faces_ijkd_=NULL;
}
bool MCGpu::init(int nx,int ny, int nz)
{	
	if(nx<=0||ny<=0||nz<=0)
		return false;
	hipMemset(d_number_record_,0,sizeof(int)*2);
	hipSetDevice(device_id_);
	if(mx*my*mz<nx*ny*nz)
	{
		mx=nx;
		my=ny;
		mz=nz;
		hipFree(d_edge_point_state_);
        hipFree(d_points_coor_);
        hipFree(d_faces_index_);
        hipFree(d_faces_ijkd_);
	    
        hipMalloc((int**)&d_edge_point_state_,sizeof(int)*nx*ny*nz*3);
		d_set_int<<<GET_BLOCKS(nx*ny*nz*3),CUDA_NUM_THREADS>>>(nx*ny*nz*3, -1, d_edge_point_state_);
	    hipMalloc((float**)&d_points_coor_,sizeof(float)*3*(int)(nx*ny*nz*12*0.05));
	    hipMalloc((int**)&d_faces_index_,sizeof(long int)*3*(int)(nx*ny*nz*5*0.05));
	    hipMalloc((int**)&d_faces_ijkd_,sizeof(int)*3*4*(int)(nx*ny*nz*5*0.05));
	}
	else
	{
		// std::vector<int> edge_point_state=std::vector<int>(nx*ny*nz*3,-1);
	 //    hipMemcpy(d_edge_point_state_,edge_point_state.data(),sizeof(int)*nx*ny*nz*3,hipMemcpyHostToDevice);
	    d_set_int<<<GET_BLOCKS(nx*ny*nz*3),CUDA_NUM_THREADS>>>(nx*ny*nz*3, -1, d_edge_point_state_);
	}
	NX=nx;
	NY=ny;
	NZ=nz;
	return true;
}
MCGpu::~MCGpu()
{
	hipFree(d_a2fVertexOffset_);
	hipFree(d_a2iEdgeConnection_);
	hipFree(d_a2fEdgeDirection_);
	hipFree(d_aiCubeEdgeFlags_);
	hipFree(d_a2iTriangleConnectionTable_);
	hipFree(d_number_record_);
	if(NX*NY*NZ>0)
	{
		hipFree(d_edge_point_state_);
		hipFree(d_points_coor_);
		hipFree(d_faces_ijkd_);
	}
}
void MCGpu::MC(float* d_sdf_, float fTargetValue)
{
	int n_thread=NX*NY*NZ;
	// std::cerr<<"1"<<std::endl;
	// CUDA_POST_KERNEL_CHECK;
    d_mc_get_mesh_on_gpu<<<GET_BLOCKS(n_thread),CUDA_NUM_THREADS>>>(n_thread,d_sdf_,d_edge_point_state_,NX,NY,NZ, fTargetValue, d_number_record_,d_a2fVertexOffset_,d_a2iEdgeConnection_,d_a2fEdgeDirection_,d_aiCubeEdgeFlags_,d_a2iTriangleConnectionTable_,d_points_coor_,d_faces_ijkd_,d_faces_index_);
 //    std::cerr<<"2"<<std::endl;
	// CUDA_POST_KERNEL_CHECK;
    hipMemcpy(number_record_.data(),d_number_record_,sizeof(int)*2,hipMemcpyDeviceToHost);
 //    std::cerr<<"3"<<std::endl;
	// CUDA_POST_KERNEL_CHECK;
    d_conver_ijkd_to_pindex<<<GET_BLOCKS(number_record_[1]),CUDA_NUM_THREADS>>>(number_record_[1],NX,NY,NZ, d_edge_point_state_, d_faces_ijkd_, d_faces_index_, d_number_record_);
 //    std::cerr<<"4"<<std::endl;
	// CUDA_POST_KERNEL_CHECK;
}
void MCGpu::scaleVertices(float xstep,float ystep,float zstep,float xmin,float ymin,float zmin)
{
	// std::cerr<<"5"<<std::endl;
	// CUDA_POST_KERNEL_CHECK;
	d_scale_vertices<<<GET_BLOCKS(number_record_[0]),CUDA_NUM_THREADS>>>(number_record_[0], xstep, ystep, zstep, xmin, ymin, zmin, d_points_coor_);
}